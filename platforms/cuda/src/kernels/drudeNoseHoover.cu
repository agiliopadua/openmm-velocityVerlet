#include "hip/hip_runtime.h"
/**
 * Calculate the center of mass velocities of each residues
 */

extern "C" __global__ void calcCOMVelocities(const mixed4 *__restrict__ velm,
                                             mixed4 *__restrict__ comVelm,
                                             const int2 *__restrict__ particlesInResidues,
                                             const int *__restrict__ particlesSortedByResId,
                                             const int *__restrict__ residuesNH) {

    // Get COM velocities
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_RESIDUES_NH; i += blockDim.x*gridDim.x) {
        int resid = residuesNH[i];
        comVelm[resid] = make_mixed4(0,0,0,0);
        mixed comMass = 0.0;
        for (int j = 0; j < particlesInResidues[resid].x; j++) {
            int index = particlesSortedByResId[particlesInResidues[resid].y + j];
            mixed4 velocity = velm[index];
            if (velocity.w != 0) {
                mixed mass = RECIP(velocity.w);
                comVelm[resid].x += velocity.x * mass;
                comVelm[resid].y += velocity.y * mass;
                comVelm[resid].z += velocity.z * mass;
                comMass += mass;
            }
        }
        comVelm[resid].w = RECIP(comMass);
        comVelm[resid].x *= comVelm[resid].w;
        comVelm[resid].y *= comVelm[resid].w;
        comVelm[resid].z *= comVelm[resid].w;

//        if (i == 0)
//            printf("residue %d has %d particles and starts at %d and vel %f,%f,%f and mass is %f \n",
//                   i, particlesInResidues[i].x, particlesInResidues[i].y,
//                   comVelm[i].x, comVelm[i].y, comVelm[i].z, RECIP(comVelm[i].w));
    }

}

/**
 * Calculate the relative velocities of each particles relative to the center of mass of each residues
 */

extern "C" __global__ void normalizeVelocities(mixed4 *__restrict__ velm,
                                               const mixed4 *__restrict__ comVelm,
                                               const int *__restrict__ particleResId,
                                               const int *__restrict__ particlesNH) {

    // Get Normalized velocities
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_PARTICLES_NH; i += blockDim.x*gridDim.x) {
        int index = particlesNH[i];
        int resid = particleResId[index];
        velm[index].x -= comVelm[resid].x;
        velm[index].y -= comVelm[resid].y;
        velm[index].z -= comVelm[resid].z;

//        if (i == 0)
//            printf("Particle: %d, Norm velocity: %f, velocity: %f, comVel: %f, mass: %f\n",
//                   i, normVelm[i].x, velm[i].x, comVelm[resid].x, RECIP(normVelm[i].w));
    }
}

/**
 * Calculate the kinetic energies of each degree of freedom.
 */

extern "C" __global__ void computeNormalizedKineticEnergies(const mixed4 *__restrict__ velm,
                                                            const mixed4 *__restrict__ comVelm,
                                                            const int *__restrict__ normalParticles,
                                                            const int2 *__restrict__ pairParticles,
                                                            double *__restrict__ kineticEnergyBuffer,
                                                            const int *__restrict__ residuesNH,
                                                            int bufferSize) {
    /**
     * the length of kineticEnergyBuff is numParticlesNH*(NUM_TEMP_GROUPS+2)
     * numThreads can be a little bit larger than numParticlesNH
     * each thread initialize (NUM_TEMP_GROUPS+2) sequential elements of kineticEnergyBuffer
     * careful to not cross the boundary of kineticEnergyBuffer
     */

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ((tid + 1) * (NUM_TEMP_GROUPS + 2) <= bufferSize) {
        for (int i = 0; i < NUM_TEMP_GROUPS + 2; i++)
            kineticEnergyBuffer[tid * (NUM_TEMP_GROUPS + 2) + i] = 0;
    }

    // Add kinetic energy of molecular motions.
    for (int i = tid; i < NUM_RESIDUES_NH; i += blockDim.x * gridDim.x) {
        int resid = residuesNH[i];
        mixed4 velocity = comVelm[resid];
        if (velocity.w != 0)
            kineticEnergyBuffer[tid * (NUM_TEMP_GROUPS + 2) + NUM_TEMP_GROUPS] +=
                    (velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z) / velocity.w;
    }

    // Add kinetic energy of ordinary particles.
    for (int i = tid; i < NUM_NORMAL_PARTICLES_NH; i += blockDim.x * gridDim.x) {
        int index = normalParticles[i];
        mixed4 velocity = velm[index];
        if (velocity.w != 0) {
            kineticEnergyBuffer[tid * (NUM_TEMP_GROUPS + 2)] +=
                    (velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z) / velocity.w;
        }
    }

    // Add kinetic energy of Drude particle pairs.
    for (int i = tid; i < NUM_PAIRS_NH; i += blockDim.x*gridDim.x) {
        int2 pair = pairParticles[i];
        mixed4 velocity1 = velm[pair.x];
        mixed4 velocity2 = velm[pair.y];
        mixed mass1 = RECIP(velocity1.w);
        mixed mass2 = RECIP(velocity2.w);
        mixed invTotalMass = RECIP(mass1+mass2);
        mixed invReducedMass = (mass1+mass2)*velocity1.w*velocity2.w;
        mixed mass1fract = invTotalMass*mass1;
        mixed mass2fract = invTotalMass*mass2;
        mixed4 cmVel = velocity1*mass1fract+velocity2*mass2fract;
        mixed4 relVel = velocity1-velocity2;

        kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)] += (cmVel.x*cmVel.x + cmVel.y*cmVel.y + cmVel.z*cmVel.z)*(mass1+mass2);
        kineticEnergyBuffer[tid*(NUM_TEMP_GROUPS+2)+NUM_TEMP_GROUPS+1] += (relVel.x*relVel.x + relVel.y*relVel.y + relVel.z*relVel.z)*RECIP(invReducedMass);
    }
}

/**
 * Sum up the kinetic energies of each degree of freedom.
 */

extern "C" __global__ void sumNormalizedKineticEnergies(double *__restrict__ kineticEnergyBuffer,
                                                        double *__restrict__ kineticEnergies,
                                                        int bufferSize) {
    /**
     * Sum kineticEnergyBuffer
     * The numThreads of this kernel equals to threadBlockSize.
     * So there is only one threadBlock for this kernel
     */
    extern __shared__ double temp[];
    unsigned int tid = threadIdx.x;

    for (unsigned int i = 0; i < NUM_TEMP_GROUPS + 2; i++)
        temp[tid * (NUM_TEMP_GROUPS + 2) + i] = 0;
    __syncthreads();

    for (unsigned int i = 0; i < NUM_TEMP_GROUPS + 2; i++) {
        for (unsigned int index = tid * (NUM_TEMP_GROUPS + 2);
             index + i < bufferSize; index += blockDim.x * (NUM_TEMP_GROUPS + 2)) {
            temp[tid * (NUM_TEMP_GROUPS + 2) + i] += kineticEnergyBuffer[index + i];
        }
    }
    __syncthreads();
    for (unsigned int i = 0; i < NUM_TEMP_GROUPS + 2; i++) {
        for (unsigned int k = blockDim.x / 2; k > 0; k >>= 1) {
            if (tid < k)
                temp[tid * (NUM_TEMP_GROUPS + 2) + i] += temp[(tid + k) * (NUM_TEMP_GROUPS + 2) + i];
            __syncthreads();
        }
    }
    if (tid == 0) {
        for (unsigned int i = 0; i < NUM_TEMP_GROUPS + 2; i++) {
            kineticEnergies[i] = temp[i];
        }
    }
}

/**
 * Perform the velocity scaling of NoseHoover thermostat.
 */

extern "C" __global__ void integrateDrudeNoseHooverVelocityScale(mixed4 *__restrict__ velm,
                                                                 const mixed4 *__restrict__ comVelm,
                                                                 const int *__restrict__ particleResId,
                                                                 const int *__restrict__ normalParticles,
                                                                 const int2 *__restrict__ pairParticles,
                                                                 const mixed *__restrict__ vscaleFactors) {

    mixed vscaleAtom = vscaleFactors[0];
    mixed vscaleCOM = vscaleFactors[1];
    mixed vscaleDrude = vscaleFactors[2];
    // Update normal particles.
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_NORMAL_PARTICLES_NH; i += blockDim.x*gridDim.x) {
        int index = normalParticles[i];
        int resid = particleResId[index];
        mixed4 velCOM = comVelm[resid];
        if (velm[index].w != 0) {
            velm[index].x = vscaleAtom*velm[index].x + vscaleCOM*velCOM.x;
            velm[index].y = vscaleAtom*velm[index].y + vscaleCOM*velCOM.y;
            velm[index].z = vscaleAtom*velm[index].z + vscaleCOM*velCOM.z;
        }
    }
    
    // Update Drude particle pairs.
    
    for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < NUM_PAIRS_NH; i += blockDim.x*gridDim.x) {
        int2 particles = pairParticles[i];
        int resid = particleResId[particles.x];
        mixed4 velAtom1 = velm[particles.x];
        mixed4 velAtom2 = velm[particles.y];
        mixed4 velCOM = comVelm[resid];
        mixed mass1 = RECIP(velAtom1.w);
        mixed mass2 = RECIP(velAtom2.w);
        mixed invTotalMass = RECIP(mass1+mass2);
        mixed mass1fract = invTotalMass*mass1;
        mixed mass2fract = invTotalMass*mass2;
        mixed4 cmVel = velAtom1*mass1fract+velAtom2*mass2fract;
        mixed4 relVel = velAtom2-velAtom1;
        cmVel.x = vscaleAtom*cmVel.x;
        cmVel.y = vscaleAtom*cmVel.y;
        cmVel.z = vscaleAtom*cmVel.z;
        relVel.x = vscaleDrude*relVel.x;
        relVel.y = vscaleDrude*relVel.y;
        relVel.z = vscaleDrude*relVel.z;
        velAtom1.x = cmVel.x-relVel.x*mass2fract + vscaleCOM*velCOM.x;
        velAtom1.y = cmVel.y-relVel.y*mass2fract + vscaleCOM*velCOM.y;
        velAtom1.z = cmVel.z-relVel.z*mass2fract + vscaleCOM*velCOM.z;
        velAtom2.x = cmVel.x+relVel.x*mass1fract + vscaleCOM*velCOM.x;
        velAtom2.y = cmVel.y+relVel.y*mass1fract + vscaleCOM*velCOM.y;
        velAtom2.z = cmVel.z+relVel.z*mass1fract + vscaleCOM*velCOM.z;
        velm[particles.x] = velAtom1;
        velm[particles.y] = velAtom2;
    }
}
